#include <hip/hip_runtime.h>
#include "mex.h"
#include <math.h>
#include <assert.h>

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>
#include <cudpp.h>
#include <jacket.h>

// Input Arguments 
#define data_IN_xi prhs[0]
#define data_IN_yi prhs[1]
#define data_IN_G prhs[2]
#define data_IN_KBlut prhs[3]
#define data_IN_nj prhs[4]

typedef unsigned int uint;

extern "C"
void gridding_reg2ir( float* xi, float* yi, 
                      float* G, size_t Gwidth, size_t Gheight,
                      float* KBlut, uint KBwidth, 
                      float* GI, size_t NGI );

err_t jktFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray *prhs[])
{ 
    float *xi, *yi, *G, *KBlut, *GI, *fKBwidth;
    uint hostKBwidth;

    TRY( jkt_mem((void **)&xi, data_IN_xi ) );
    TRY( jkt_mem((void **)&yi, data_IN_yi ) );
    TRY( jkt_mem((void **)&G, data_IN_G ) );
    TRY( jkt_mem((void **)&KBlut, data_IN_KBlut ) );

    TRY( jkt_mem_host((void **)&fKBwidth, data_IN_nj ) );
    hostKBwidth = (uint)(fKBwidth[0]);

    // Output size is length of input xi and yi
    const mwSize *dimsxi, *dimsG;
    int in_dimxi, in_dimG;
    mwSize nxi, nG;
    uint Gwidth, Gheight;

    in_dimxi = jkt_dims( data_IN_xi, &dimsxi );
    nxi = jkt_numel(data_IN_xi);

    in_dimG = jkt_dims( data_IN_G, &dimsG );
    Gheight = dimsG[0];
    Gwidth = dimsG[1];
    nG = jkt_numel(data_IN_G);

    if( 0 ) {
        FILE* fp = fopen( "/home/jpien/ndump.txt", "a" );
        if( fp ) {
            for( int iii=0; iii < in_dimxi; iii++ ) {
                fprintf( fp, "Size of xi: [%d]=%d\n", iii, dimsxi[iii] );
            }
            fprintf( fp, "Size of xi: %d\n", nxi );
            for( int iii=0; iii < in_dimG; iii++ ) {
                fprintf( fp, "Size of G: [%d]=%d\n", iii, dimsG[iii] );
            }
            fprintf( fp, "Size of G: %d\n", nG );
            fprintf( fp, "Width of kernel = %d\n", hostKBwidth );
            fflush( fp );
        }
        fclose( fp );
    }

    // Error checking
    if( jkt_complex( data_IN_xi ) ||
        jkt_complex( data_IN_yi ) || 
        jkt_complex( data_IN_G ) ||
        jkt_complex( data_IN_KBlut) ) {
        return err( "Input data must be real, non-complex\n" );
    }
    if( (Gheight & (Gheight-1) != 0) ||
        (Gwidth & (Gwidth-1) != 0) ) {
        return err( "G must have dimensions of power of two\n" );
    }
    if( jkt_numel(data_IN_KBlut) != 
        ((hostKBwidth+1)*20+1)*((hostKBwidth+1)*20+1) ) {
        return err( "KBlut size does not match KBwidth\n" );
    }
    if( jkt_numel(data_IN_xi) != jkt_numel(data_IN_yi) ) {
        return err( "Size of xi and yi is not the same\n" );
    }        
    
    // Output is same number of elements as input irregular sample points
    mxArray *data_OUT_GI = plhs[0] = 
        jkt_new_array( in_dimxi, dimsxi, mxSINGLE_CLASS, 0 );
    TRY( jkt_mem((void **)&GI,  data_OUT_GI) );

    // Error checking
    if( false ) {
        return err( "Invalid input parameters\n" );
    }

    // Call CUDA
    gridding_reg2ir( xi, yi, G, Gwidth, Gheight, 
                     KBlut, hostKBwidth, GI, nxi );

    return errNone;
}
