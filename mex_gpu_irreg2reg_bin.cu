#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mex.h"
#include <math.h>
#include <assert.h>

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>
#include <cudpp.h>
#include <jacket.h>

// Input Arguments 
#define data_IN_xi prhs[0]
#define data_IN_yi prhs[1]
#define data_IN_GI prhs[2]
#define data_IN_KBlut prhs[3]
#define data_IN_nj prhs[4]
#define data_IN_siz prhs[5]
#define data_IN_binidx prhs[6]
#define data_IN_binstartidx prhs[7]
#define data_IN_binlength prhs[8]

typedef unsigned int uint;

extern "C"
void gridding_irreg2reg_bin( float* xi, float* yi, 
                             float* GI, size_t NGI,
                             float* KBlut, uint KBwidth, 
                             float* G, size_t Gwidth, size_t Gheight,
                             uint* binidx, uint* binstartidx, 
                             uint* binlength, uint tiledim );

err_t jktFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray *prhs[])
{ 
    float *xi, *yi, *G, *Gsiz, *KBlut, *GI, *fKBwidth;
    uint hostKBwidth, Gwidth, Gheight;
    uint *binidx, *binstartidx, *binlength;

    TRY( jkt_mem((void **)&xi, data_IN_xi ) );
    TRY( jkt_mem((void **)&yi, data_IN_yi ) );
    TRY( jkt_mem((void **)&GI, data_IN_GI ) );
    TRY( jkt_mem((void **)&KBlut, data_IN_KBlut ) );
    TRY( jkt_mem((void **)&binidx, data_IN_binidx ) );
    TRY( jkt_mem((void **)&binstartidx, data_IN_binstartidx ) );
    TRY( jkt_mem((void **)&binlength, data_IN_binlength ) );

    TRY( jkt_mem_host((void **)&fKBwidth, data_IN_nj ) );
    hostKBwidth = (uint)(fKBwidth[0]);

    TRY( jkt_mem_host((void **)&Gsiz, data_IN_siz ) );
    Gheight = (uint)(Gsiz[0]);
    Gwidth = (uint)(Gsiz[1]);

    mwSize NGI, len;

    NGI = jkt_numel(data_IN_GI);

    // Compute tile length
    len = jkt_numel(data_IN_binstartidx);
    len = Gwidth * Gheight / len;
    len = sqrt(len);
    
    // Output 
    mxArray *data_OUT_G = plhs[0] = 
        jkt_new( Gheight, Gwidth, mxSINGLE_CLASS, 0 );
    TRY( jkt_mem((void **)&G,  data_OUT_G) );

    // Error checking
    if( jkt_complex( data_IN_xi ) ||
        jkt_complex( data_IN_yi ) || 
        jkt_complex( data_IN_GI ) ||
        jkt_complex( data_IN_KBlut) ) {
        return err( "Input data must be real, non-complex\n" );
    }
    if( (Gheight & (Gheight-1) != 0) ||
        (Gwidth & (Gwidth-1) != 0) ) {
        return err( "G must have dimensions of power of two\n" );
    }
    if( jkt_numel(data_IN_KBlut) != 
        ((hostKBwidth+1)*20+1)*((hostKBwidth+1)*20+1) ) {
        return err( "KBlut size does not match KBwidth\n" );
    }
    if( jkt_numel(data_IN_xi) != jkt_numel(data_IN_yi) ||
        jkt_numel(data_IN_xi) != jkt_numel(data_IN_GI) ) {
        return err( "Size of xi, yi and/or GI is not the same\n" );
    }  
    if( jkt_numel(data_IN_binstartidx) != jkt_numel(data_IN_binlength) ) {
        return err( "Size of binstartidx and binlength is not the same\n" );
    }

    // Call CUDA
    gridding_irreg2reg_bin( xi, yi, 
                            GI, NGI,
                            KBlut, hostKBwidth, 
                            G, Gwidth, Gheight,
                            binidx, binstartidx, binlength, len );

    return errNone;
}
