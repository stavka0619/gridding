#include <hip/hip_runtime.h>
#include "mex.h"
#include <math.h>
#include <assert.h>

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>
#include <cudpp.h>
#include <jacket.h>

// Input Arguments 
#define data_IN_xi prhs[0]
#define data_IN_yi prhs[1]
#define data_IN_GI prhs[2]
#define data_IN_KBlut prhs[3]
#define data_IN_nj prhs[4]
#define data_IN_siz prhs[5]

typedef unsigned int uint;

extern "C"
void gridding_irreg2reg( float* xi, float* yi, 
                         float* GI, size_t NGI,
                         float* KBlut, uint KBwidth, 
                         float* G, size_t Gwidth, size_t Gheight );

err_t jktFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray *prhs[])
{ 
    float *xi, *yi, *G, *Gsiz, *KBlut, *GI, *fKBwidth;
    uint hostKBwidth, Gwidth, Gheight;

    TRY( jkt_mem((void **)&xi, data_IN_xi ) );
    TRY( jkt_mem((void **)&yi, data_IN_yi ) );
    TRY( jkt_mem((void **)&GI, data_IN_GI ) );
    TRY( jkt_mem((void **)&KBlut, data_IN_KBlut ) );

    TRY( jkt_mem_host((void **)&fKBwidth, data_IN_nj ) );
    hostKBwidth = (uint)(fKBwidth[0]);

    TRY( jkt_mem_host((void **)&Gsiz, data_IN_siz ) );
    Gheight = (uint)(Gsiz[0]);
    Gwidth = (uint)(Gsiz[1]);

    mwSize NGI;

    NGI = jkt_numel(data_IN_GI);
    
    // Output 
    mxArray *data_OUT_G = plhs[0] = 
        jkt_new( Gheight, Gwidth, mxSINGLE_CLASS, 0 );
    TRY( jkt_mem((void **)&G,  data_OUT_G) );

    // Error checking
    if( jkt_complex( data_IN_xi ) ||
        jkt_complex( data_IN_yi ) || 
        jkt_complex( data_IN_GI ) ||
        jkt_complex( data_IN_KBlut) ) {
        return err( "Input data must be real, non-complex\n" );
    }
    if( (Gheight & (Gheight-1) != 0) ||
        (Gwidth & (Gwidth-1) != 0) ) {
        return err( "G must have dimensions of power of two\n" );
    }
    if( jkt_numel(data_IN_KBlut) != 
        ((hostKBwidth+1)*20+1)*((hostKBwidth+1)*20+1) ) {
        return err( "KBlut size does not match KBwidth\n" );
    }
    if( jkt_numel(data_IN_xi) != jkt_numel(data_IN_yi) ||
        jkt_numel(data_IN_xi) != jkt_numel(data_IN_GI) ) {
        return err( "Size of xi, yi and/or GI is not the same\n" );
    }        

    // Call CUDA
    gridding_irreg2reg( xi, yi, 
                        GI, NGI,
                        KBlut, hostKBwidth, 
                        G, Gwidth, Gheight );

    return errNone;
}
