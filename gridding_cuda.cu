#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <cutil_inline.h>
#include <hip/hip_runtime.h>

#include "utilCuda.h"

__device__ float kSample( mxType* KBlut, size_t KBwidth,
                         float x, float y )
{
    // Linear interpolate
    int offset = ((KBwidth+1)*10)+0; // Kernel radius is 1 larger than width
    int pitch = (((KBwidth+1)*2)*10)+1; 
    int x0 = floorf(x*10.0) + offset;
    int x1 = ceilf(x*10.0) + offset;
    int y0 = floorf(y*10.0) + offset;
    int y1 = ceilf(y*10.0) + offset;
    float xd = x*10.0 - floorf(x*10.0);
    float yd = y*10.0 - floorf(y*10.0);

//     float nw = KBlut[ (x0*pitch + y0) ];
//     float ne = KBlut[ (x1*pitch + y0) ];
//     float sw = KBlut[ (x0*pitch + y1) ];
//     float se = KBlut[ (x1*pitch + y1) ];

    float nw = KBlut[ (y0*pitch + x0) ];
    float ne = KBlut[ (y0*pitch + x1) ];
    float sw = KBlut[ (y1*pitch + x0) ];
    float se = KBlut[ (y1*pitch + x1) ];

    float nw_ne = (nw * (1-xd)) + (ne * xd);
    float sw_se = (sw * (1-xd)) + (se * xd);
    float val = (nw_ne * (1-yd)) + (sw_se * yd);
    
    return val;
}

__global__ void reg2ir_cuda( mxType* xi, mxType* yi, 
                             mxType* G, size_t Gwidth, size_t Gheight,
                             mxType* KBlut, size_t KBwidth, 
                             mxType* GI, size_t NGI )
{
    volatile int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
    float fkbw = (float)KBwidth;
    float invGwidth = 1.0/(float)Gwidth;
    float invGheight = 1.0/(float)Gheight;

    if( tid < NGI ) {
        // Figure out sample pattern
        float xx = xi[tid]-1;
        float yy = yi[tid]-1;
        float nearx = rintf( xx );
        float neary = rintf( yy );
        float distxstart = xx - (nearx - fkbw);
        float distystart = yy - (neary - fkbw);
        float xstart = nearx - fkbw;
        float ystart = neary - fkbw;
        float accum = 0;

        for( int ww=0; ww < KBwidth*2+1; ww++ ) {
            float xnorm = (xstart+(float)ww)*invGwidth;
            int xcoord = (xnorm - floorf(xnorm)) * (float)Gwidth; 

            for( int hh=0; hh < KBwidth*2+1; hh++ ) {
                float ynorm = (ystart+(float)hh)*invGheight;
                int ycoord = (ynorm - floorf(ynorm)) * (float)Gheight;
                float Gsample = G[xcoord * Gheight + ycoord];
                float kernelSample = kSample( KBlut, KBwidth, 
                                              distxstart - ww,
                                              distystart - hh );
                accum += Gsample * kernelSample;
            }
        }
        GI[tid] = accum;
    }
}   

extern "C"
void gridding_reg2ir( mxType* xi, mxType* yi, 
                      mxType* G, size_t Gwidth, size_t Gheight,
                      mxType* KBlut, size_t KBwidth, 
                      mxType* GI, size_t NGI )
{
    uint threadsPerBlock = 512; 
    uint blocksPerGrid = (NGI + threadsPerBlock - 1) / threadsPerBlock;
    
    reg2ir_cuda
        <<<blocksPerGrid, threadsPerBlock>>>
        ( xi, yi, G, Gwidth, Gheight,
          KBlut, KBwidth,
          GI, NGI );
    
    hipError_t e = hipGetLastError();
    if( e != hipSuccess ) {
        assert( e == hipSuccess );
    }
}

typedef struct {
    float x;
    float y;
    float gi;
} ircache;
#define IRCACHE_SZ 1024
#define GMEMORYREAD_SZ 64 // 64bytes for half warp coleasced read
#define THREADS_PERBLOCK 256
#define TILE_DIM 16

/******************************************************************************/
__device__ float inblockregion( float x, float y,
                                float nwx, float nwy, float sex, float sey ) {
   
    if( (x > nwx && y > nwy &&
         x < sex && y < sey) ) {
        return true;
    } else {
        return false;
    }
}
__device__ int finblockregion( float x, float y,
                               float nwx, float nwy, float sex, float sey ) {
   
    if( (x > nwx && y > nwy &&
         x < sex && y < sey) ) {
        return 1;
    } else {
        return 0;
    }
}

__global__ void irreg2reg_cuda( mxType* xi, mxType* yi,
                                mxType* GI, size_t NGI,
                                mxType* KBlut, size_t KBwidth,  
                                mxType* G, size_t Gwidth, size_t Gheight )
{
    int id = threadIdx.x;
    mxType sample=0;

    __shared__ int nir;
    __shared__ ircache ir[IRCACHE_SZ];
    __shared__ ircache tmpir[ THREADS_PERBLOCK ];
    __shared__ short tmpir_inregion[ THREADS_PERBLOCK ];
    float blkx;
    float blky;

    // First thread initializes some values
    if( id == 0 ) {
        // Number of irregular points i the cache. Inits to zero
        nir = 0;
    }

    // Block coordinates within the grid
    // There are nBlocks in grid width
    float nBlocksHigh = Gheight/TILE_DIM; 
    blkx = floorf( blockIdx.x/nBlocksHigh );
    blky = blockIdx.x - (blkx * nBlocksHigh);

    // The block area we care about within the whole grid
    int nwy = blky * TILE_DIM; 
    int nwx = blkx * TILE_DIM;
    int sey = nwy + TILE_DIM-1;
    int sex = nwx + TILE_DIM-1;
        
    // Add filter window area
    nwy -= (KBwidth+1);
    nwx -= (KBwidth+1);
    sey += (KBwidth+1);
    sex += (KBwidth+1);

    // All threads read irregular points from global to 
    // optimize coalesced reads in "chunks". Then 
    // thread 0 is going to cache all irregular points within
    // region of interest in shared memory.
    for( int iii=0; iii < (int)(NGI/blockDim.x); iii++ ) {
        int idx = (iii*blockDim.x) + id;
        int ntmpir = 0;
        tmpir_inregion[id] = 0.0;

        if( idx < NGI ) {
            tmpir[ id ].x = xi[idx] - 1; // Matlab indices
            tmpir[ id ].y = yi[idx] - 1;
            tmpir[ id ].gi = GI[idx];

            // See if we care about this ir?
            tmpir_inregion[id] = finblockregion( tmpir[id].x, tmpir[id].y,
                                                 nwx, nwy, sex, sey );
        }

        __syncthreads();

        // Count the number of ir's in region
        for( uint s=blockDim.x/2; s > 0; s>>=1 ) {
            if( id < s ) {
                tmpir_inregion[id] += tmpir_inregion[id + s];
            }
            __syncthreads();
        }

        // Thread zero writes away total number of ir's we care about
        if( id == 0 ) {
            ntmpir = tmpir_inregion[id];
        }

        // Retag the irs we care about
        if( idx < NGI ) {
            tmpir_inregion[id] = finblockregion( tmpir[id].x, tmpir[id].y,
                                                 nwx, nwy, sex, sey );
        }            

        __syncthreads();

        // Thread 0 is going to cache all irregular points
        if( id == 0 ) {
            // Too many for cache
            if( ntmpir > IRCACHE_SZ - nir ) {
                // Crap, no more cache room.  Let's stop looking
                // We are now on slow path
                nir = -1;
                iii = NGI;
            } else {
                
                // If the irreg point is in the area we care about,
                // then cache it away
                for( int jj = 0; 
                     jj < blockDim.x && 
                         (jj+(iii*blockDim.x)) < NGI; jj++ ) {
                    if( tmpir_inregion[jj] == 1 ) {
                    
                        // Cache it away
                        ir[nir] = tmpir[jj];
                        nir++;                    
                    }
                }
            }
        }

        __syncthreads();
    } 

    // Coord within tile or block (stored col major)
    int posx = floorf(id/TILE_DIM);
    int posy = id - (posx * TILE_DIM);

    // Coord within whole grid
    int gridy = posy + (blky * TILE_DIM); 
    int gridx = posx + (blkx * TILE_DIM);

    // Add filter window area
    sey = gridy + (KBwidth+1);
    sex = gridx + (KBwidth+1);
    nwy = gridy - (KBwidth+1);
    nwx = gridx - (KBwidth+1);

    // If we were able to cache all the points we care about then have 
    // each thread compute its output
    if( nir >= 0 ) {
        for( int iii=0; iii < nir; iii++ ) {
            if( inblockregion( ir[iii].x, ir[iii].y,
                               (float)nwx, (float)nwy, 
                               (float)sex, (float)sey ) ) {
                sample += ir[iii].gi * kSample( KBlut, KBwidth, 
                                                ir[iii].x - gridx, 
                                                ir[iii].y - gridy );
            }
        }
    } else {
        // Oh crap, we couldn't cache everything so we need to take the
        // REALLY slow path and read each irreg point from global
        // memory
        for( int iii=0; iii < NGI; iii++ ) {
            float irx = xi[iii] - 1;
            float iry = yi[iii] - 1;
            if( inblockregion( irx, iry,
                               (float)nwx, (float)nwy, 
                               (float)sex, (float)sey ) ) {
                sample += GI[iii] * kSample( KBlut, KBwidth, 
                                             irx - gridx, iry - gridy );
            }
        } 
    }

    // Write out the output
    G[gridx * Gheight + gridy ] = sample;
}   

extern "C"
void gridding_irreg2reg( mxType* xi, mxType* yi, 
                         mxType* GI, size_t NGI,
                         mxType* KBlut, uint KBwidth, 
                         mxType* G, size_t Gwidth, size_t Gheight )
{
    // Want to divide work so each thread works on an output
    // coord.  Each block is a 16x16 tile.
    uint threadsPerBlock = THREADS_PERBLOCK; 
    uint blocksPerGrid = 
        ((Gwidth*Gheight) + threadsPerBlock - 1) / threadsPerBlock;

    irreg2reg_cuda
        <<<blocksPerGrid, threadsPerBlock>>>
        ( xi, yi, GI, NGI, KBlut, KBwidth, 
          G, Gwidth, Gheight );
    
    hipError_t e = hipGetLastError();
    if( e != hipSuccess ) {
        fprintf( stderr, "Error running kernel <%d>\n", e );
        assert( e == hipSuccess );
    }
}

/*****************************************************************************/
__global__ void irreg2reg_cuda_bin( mxType* xi, mxType* yi,
                                    mxType* GI, size_t NGI,
                                    mxType* KBlut, size_t KBwidth,  
                                    mxType* G, size_t Gwidth, size_t Gheight,
                                    uint* binidx, uint* binstartidx, 
                                    uint* binlength )
{
    int id = threadIdx.x;
    mxType sample=0;

    int nir;
    __shared__ ircache ir[IRCACHE_SZ];
    float blkx;
    float blky;

    // First thread initializes some values
    if( id == 0 ) {
        // Number of irregular points i the cache. Inits to zero
        nir = 0;
    }

    // Load irreg into cache
    uint numbin = binlength[blockIdx.x];
    uint startidx = binstartidx[blockIdx.x]-1;
    if( numbin <= IRCACHE_SZ ) {
        for( int iii=0; iii <= (numbin/blockDim.x); iii++ ) {
            int idx = (iii*blockDim.x) + id;
            if( idx < numbin ) {
                uint bidx = (binidx[startidx + idx]) - 1;
                ir[idx].x = xi[bidx]-1;
                ir[idx].y = yi[bidx]-1;
                ir[idx].gi = GI[bidx];
            }
        }
        nir = numbin;
    } else {
        // Can't load it into cache
        nir = -1;
    }

    __syncthreads();

    // Block coordinates within the grid
    // There are nBlocks in grid width
    float nBlocksHigh = Gheight/TILE_DIM; 
    blkx = floorf( blockIdx.x/nBlocksHigh );
    blky = blockIdx.x - (blkx * nBlocksHigh);

    // The block area we care about within the whole grid
    int nwy = blky * TILE_DIM; 
    int nwx = blkx * TILE_DIM;
    int sey = nwy + TILE_DIM-1;
    int sex = nwx + TILE_DIM-1;
        
    // Add filter window area
    nwy -= (KBwidth+1);
    nwx -= (KBwidth+1);
    sey += (KBwidth+1);
    sex += (KBwidth+1);

    // Coord within tile or block (stored col major)
    int posx = floorf(id/TILE_DIM);
    int posy = id - (posx * TILE_DIM);

    // Coord within whole grid
    int gridy = posy + (blky * TILE_DIM); 
    int gridx = posx + (blkx * TILE_DIM);

    // Add filter window area
    sey = gridy + (KBwidth+1);
    sex = gridx + (KBwidth+1);
    nwy = gridy - (KBwidth+1);
    nwx = gridx - (KBwidth+1);

    // If we were able to cache all the points we care about then have 
    // each thread compute its output
    if( nir >= 0 ) {
        for( int iii=0; iii < nir; iii++ ) {
            if( inblockregion( ir[iii].x, ir[iii].y,
                               (float)nwx, (float)nwy, 
                               (float)sex, (float)sey ) ) {
                sample += ir[iii].gi * kSample( KBlut, KBwidth, 
                                                ir[iii].x - gridx, 
                                                ir[iii].y - gridy ); 
            }
        }
    } else {
        // Oh crap, we couldn't cache everything so we need to take the
        // REALLY slow path and read each irreg point from global
        // memory
        for( int iii=0; iii < numbin; iii++ ) {
            uint bidx = (binidx[startidx + iii]) - 1;
            float irx = xi[bidx] - 1;
            float iry = yi[bidx] - 1;
            if( inblockregion( irx, iry,
                               (float)nwx, (float)nwy, 
                               (float)sex, (float)sey ) ) {
                sample += GI[bidx] * kSample( KBlut, KBwidth, 
                                              irx - gridx, iry - gridy );
            }
        } 
    }

    // Write out the output
    G[gridx * Gheight + gridy ] = sample;
}     

extern "C"
void gridding_irreg2reg_bin( mxType* xi, mxType* yi, 
                             mxType* GI, size_t NGI,
                             mxType* KBlut, uint KBwidth, 
                             mxType* G, size_t Gwidth, size_t Gheight,
                             uint* binidx, uint* binstartidx, 
                             uint* binlength, uint tiledim )
{
    // Want to divide work so each thread works on an output
    // coord.  Each block is a 16x16 tile.
    uint threadsPerBlock = THREADS_PERBLOCK; 
    uint blocksPerGrid = 
        ((Gwidth*Gheight) + threadsPerBlock - 1) / threadsPerBlock;

    assert( tiledim == TILE_DIM );
    
    irreg2reg_cuda_bin
        <<<blocksPerGrid, threadsPerBlock>>>
        ( xi, yi, GI, NGI, KBlut, KBwidth, 
          G, Gwidth, Gheight, binidx, binstartidx, binlength );
    
    hipError_t e = hipGetLastError();
    if( e != hipSuccess ) {
        assert( e == hipSuccess );
    }

}
